//
// Created by nasser on 06/07/23.
//

#include "Moo_Funcs.cuh"

FunctionEvaluator::FunctionEvaluator(KernelFunction kernelFunc) : kernel_function(kernelFunc)
{

}

FunctionEvaluator::~FunctionEvaluator()
{
    hipFree(d_X);
    hipFree(d_Y);
}

void FunctionEvaluator::evaluate(int **X, int *Y, int numElements)
{
    hipMalloc((void**)&d_X, sizeof(int*) * numElements);
    hipMemcpy(d_X, X, sizeof(int*) * numElements, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_Y, sizeof(int) * numElements);

    int blockSize = 256;
    int numBlocks = (numElements + blockSize - 1) / blockSize;

    kernel_function<<<numBlocks, blockSize>>>(d_X, d_Y, numElements);

    hipMemcpy(Y, d_Y, sizeof(int) * numElements, hipMemcpyDeviceToHost);
}
