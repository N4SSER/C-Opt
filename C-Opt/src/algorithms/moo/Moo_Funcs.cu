//
// Created by nasser on 06/07/23.
//

#include "Moo_Funcs.cuh"

MOO_func::MOO_func(KernelFunction kernelFunc) : kernel_function(kernelFunc)
{

}

MOO_func::~MOO_func()
{
    hipFree(d_X);
    hipFree(d_Y);
}

void MOO_func::evaluate(int **X, int *Y, int numElements)
{
    hipMalloc((void**)&d_X, sizeof(int*) * numElements);
    hipMemcpy(d_X, X, sizeof(int*) * numElements, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_Y, sizeof(int) * numElements);

    int blockSize = 256;
    int numBlocks = (numElements + blockSize - 1) / blockSize;

    kernel_function<<<numBlocks, blockSize>>>(d_X, d_Y, numElements);

    hipMemcpy(Y, d_Y, sizeof(int) * numElements, hipMemcpyDeviceToHost);
}
